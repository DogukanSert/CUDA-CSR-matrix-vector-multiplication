
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>
#include <float.h>

void read_matrix(int **r_ptr, int** c_ind,float** v, char*fname,int* r_count,int* v_count){	
	FILE * file;
    	if ((file = fopen(fname, "r+")) == NULL)
	{
	    printf("ERROR: file open failed\n");
	    return;
	}
	
	int column_count,row_count,values_count;
	fscanf(file, "%d %d %d\n",&row_count,&column_count,&values_count);
	*r_count = row_count;
	*v_count = values_count;
	int i;
	int *row_ptr =(int*) malloc((row_count+1) * sizeof(int));
	int *col_ind =(int*) malloc(values_count * sizeof(int));
	for(i=0; i<values_count; i++){
		col_ind[i] = -1;
	}
	float *values =(float*) malloc(values_count * sizeof(float));
	int row,column;
	float value;
	while (1) {
		int ret = fscanf(file, "%d %d %f\n",&row,&column,&value);
		column --;
		row --;
		if(ret == 3){
			row_ptr[row]++;
		} else if(ret == EOF) {
		   	break;
		} else {
		    	printf("No match.\n");
		}
	}
    	rewind(file);
    	int index = 0;
    	int val = 0;
	for(i = 0; i<row_count;i++){
		val = row_ptr[i];
		row_ptr[i] = index;
		index += val;
	}
	row_ptr[row_count] = values_count;
	fscanf(file, "%d %d %d\n",&row_count,&column_count,&values_count);
	i = 0;
	while (1) {
		int ret = fscanf(file, "%d %d %f\n",&row,&column,&value);
		column --;
		row --;
		if(ret == 3){
			while(col_ind[i+row_ptr[row]] != -1){ i++;}
			col_ind[i+row_ptr[row]] = column;
			values[i+row_ptr[row]] = value;
			i=0;
		} else if(ret == EOF) {
		   	break;
		} else {
		    	printf("No match.\n");
		}
	}
    	fclose(file);
    	*r_ptr = row_ptr;
    	*c_ind = col_ind;
    	*v = values;
}

__global__ void mat_vector_multiply(const int num_rows,const int *ptr,const int *indices,const float *data,
				const float *x, float* y){
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	int row_start, row_end;
	float dot;
	if(row < num_rows){
		dot = 0;
		row_start = ptr[row];
		row_end = ptr[row + 1];
		for(i = row_start; i < row_end; i++){
			dot+= data[i] * x[indices[i]];
		}
	}
	y[row] += dot;
}

int main (int argc, char* argv[]){
	if ( argc != 5){
		printf( "Incorrect usage");
	}
	else{
		int* row_ptr;
		int* col_ind;
		float* values;
		int r_count, v_count, i, k;
		int thread_num = atoi(argv[1]);
		int repetitions = atoi(argv[2]);
		int mode = atoi(argv[3]);
		char* fname = argv[4];
		read_matrix(&row_ptr, &col_ind, &values, fname, &r_count, &v_count);
		float* x =(float*) malloc(r_count* sizeof(float));
		float* y =(float*) calloc(r_count, sizeof(float));
		for(i = 0; i<r_count;i++){
		    	x[i]= 1.0;
		}
		if(mode == 1){
			fprintf(stdout,"Initial Matrix\n");
			for(i = 0; i<r_count;i++){
		    		if(i+1 < r_count){
		    			for(k = row_ptr[i]; k < row_ptr[i+1];k++){
		    				fprintf(stdout,"%d %d %.10f\n",i+1,col_ind[k]+1,values[k]);
		    			}
		    		}	
		    	}
		    	fprintf(stdout,"Initial Vector\n");
		    	for(i = 0; i<r_count;i++){
		    		fprintf(stdout,"%f\n",x[i]);
		    	}
	  	}
		int *d_row_ptr, *d_col_ind;
		float *d_values, *d_x, *d_y;
		hipMalloc(&d_row_ptr, r_count*sizeof(int));
		hipMalloc(&d_col_ind, v_count*sizeof(int));
		hipMalloc(&d_values, v_count*sizeof(int));
		hipMalloc(&d_x, r_count*sizeof(float));
		hipMalloc(&d_y, r_count*sizeof(float));
		hipMemcpy(d_row_ptr, row_ptr, r_count*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_col_ind, col_ind, v_count*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_values, values, v_count*sizeof(int), hipMemcpyHostToDevice);
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);	  	
	  	// device inputs;
	  	hipEventRecord(start);
	  	for(k = 0; k<repetitions; k++){
			hipMemcpy(d_x, x, r_count*sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_y, y, r_count*sizeof(float), hipMemcpyHostToDevice);
		
			// kernel call
			int blocksize = 64;
			int blocknum = ceil(r_count/blocksize); //number of threads fixed and equal to row count
			mat_vector_multiply <<<blocknum, blocksize>>>(r_count, d_row_ptr, d_col_ind, d_values, d_x, d_y);
		
			hipMemcpy(y, d_y, r_count*sizeof(float), hipMemcpyDeviceToHost);
			for(i = 0; i<r_count;i++){
				x[i] = y[i];
			    	y[i]= 0.0;
			}
		}
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		int count = 0;
		if(mode == 1){
			fprintf(stdout,"Resulting Vector\n");
		    	for(i = 0; i<r_count;i++){
		    		if(x[i] != 0){
		    		fprintf(stdout,"%.10f\n",x[i]);
		    		count++;
		    		}
		    	}
		    	fprintf(stdout,"count = %d\n", count);
		}
		fprintf(stdout,"time = %f\n", milliseconds);		
	}
}
